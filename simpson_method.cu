#include <stdint.h>    /* for uint64 definition */
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define MILLION 1000000
#define BILLION 1000000000L

#define BLOCK_SIZE 512

#define EPS 0.01


// подынтегральная функция
__host__ __device__ double function(double x)
{   
    return exp(-x*x);
    // return 8 + 2*x - x*x;
}

// ядро - метод Симпсона
__global__ void simpson_kernel(double a, int m, double h, double *results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < m)
    {
        double x_2i = a + 2*idx * h;
        double x_2i_1 = a + (2*idx + 1) * h;
        double x_2i_2 = a + (2*idx + 2) * h;

        double y_2i = function(x_2i);
        double y_2i_1 = function(x_2i_1);
        double y_2i_2 = function(x_2i_2);

        results[idx] = (h / 3) * (y_2i + 4*y_2i_1 + y_2i_2);
    }
}

// запуск на GPU
uint64_t run_in_gpu(double a, int m, double h)
{
    float dt_gpu;
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    // выделение памяти на CPU
    int num_bytes = sizeof(double) * m; 
    double *results = (double *)malloc(num_bytes);

    // указатели на память на видеокарте
    double *results_gpu = NULL;

    // старт события
    hipEventRecord(start_gpu, 0);

    // выделение памяти на видеокарте
    hipMalloc((void**)&results_gpu, num_bytes);

    // создание конфигурации потоков и блоков
    dim3 blockSize = dim3(BLOCK_SIZE, 1);
    dim3 numBlocks = dim3(m / blockSize.x, 1);

    // вызов ядра для метода Симпсона
    simpson_kernel<<<numBlocks, blockSize>>>(a, m, h, results_gpu);

    // копирование данных (результат вычислений) с GPU на CPU
    hipMemcpy(results, results_gpu, num_bytes, hipMemcpyDeviceToHost);

    // суммирование
    double integral = 0.0;
    for (int i = 0; i < m; ++i) {
        integral += results[i];
    }
    printf("I = %f\n", integral);

    // освобождение видеопамяти
    hipFree(results_gpu);

    // окончание события и измерение времени выполнения в ns
    hipEventRecord(stop_gpu, 0);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&dt_gpu, start_gpu, stop_gpu);
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);
    dt_gpu *= MILLION;

    // освобождение памяти на CPU
    free(results);

    return dt_gpu;
}

// запуск на CPU
uint64_t run_in_process(double a, int m, double h)
{
    clock_t t0 = clock();
    struct timespec start, stop;
    clock_gettime(CLOCK_MONOTONIC, &start);

    double integral = 0;
    double x_2i, x_2i_1, x_2i_2;
    double y_2i, y_2i_1, y_2i_2;

    // вычисление значения для каждого из M подотрезков
    for (int i = 0; i < m; ++i) 
    {
        if (i < m)
        {
            x_2i = a + 2*i * h;
            x_2i_1 = a + (2*i + 1) * h;
            x_2i_2 = a + (2*i + 2) * h;

            y_2i = function(x_2i);
            y_2i_1 = function(x_2i_1);
            y_2i_2 = function(x_2i_2);

            integral += (h / 3) * (y_2i + 4*y_2i_1 + y_2i_2);
        }
    }
    printf("I = %f\n", integral);

    clock_gettime(CLOCK_MONOTONIC, &stop);
    uint64_t diff = BILLION * (stop.tv_sec - start.tv_sec) + stop.tv_nsec - start.tv_nsec;
    return diff;
}

int main()
{
    // пределы интегрирования
    double a = -100000;
    double b = 100000; 

    int n_min = (int)((b-a) / EPS);
    int n = (n_min / (BLOCK_SIZE*2)) * (BLOCK_SIZE*2) + (n_min % (BLOCK_SIZE*2)) * (BLOCK_SIZE*2);
    int m = (int)(n / 2);
    double h = (b-a) / n;

    // вычисления на CPU
    uint64_t dt_process = run_in_process(a, m, h);
    printf("Time on CPU: %f s \n", (double) dt_process / BILLION);

    // вычисления на GPU
    uint64_t dt_gpu = run_in_gpu(a, m, h);
    printf("Time on GPU: %f s \n", (double) dt_gpu / BILLION);

    // коэффициент
    printf("Ratio: %f \n", (double)dt_process / (double)dt_gpu);
}